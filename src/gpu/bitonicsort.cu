#include "bitonicsort.cuh"
#include "hip/hip_runtime.h"


#define getLastCudaError(msg)      getLastCudaError (msg, __FILE__, __LINE__)

inline void getLastCudaError_(const char *errorMessage, const char *file, const int line)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                file, line, errorMessage, (int)err, hipGetErrorString(err));
        fflush(stderr);
        //DEVICE_RESET
        //        exit(EXIT_FAILURE);
    }
}




namespace gpu_sort{

    void bitonicSort(std::vector<double>& arr){

    }
}